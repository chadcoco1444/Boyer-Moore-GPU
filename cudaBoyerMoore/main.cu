#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <stdbool.h>
#include <time.h>

//#define SMALL_PRINT
#define CPU
#define GPU

__device__ int d_matCnt;

#include "../printMatch.c"
#include "../boyerMoore.c"
#include <hip/hip_runtime.h>


int main(int argc, char const *argv[]){

	int sourceSize, patternSize;

	FILE *source;
	FILE *pattern;
	source = fopen("../../source128MB.txt", "rb");
	pattern = fopen("../../pattern.txt", "rb");
	
	if(!source && !pattern){
		fprintf(stderr, ANSI_COLOR_RED"Cannot open the file\n"ANSI_COLOR_RESET);
		exit(1);
	}

	char *sBuffer;	
	char *pBuffer;
	
	fseek(source, 0, SEEK_END);
    sourceSize = ftell(source);
    rewind(source);

    fseek(pattern, 0, SEEK_END);
    patternSize = ftell(pattern);
    rewind(pattern);

    sBuffer = (char *)calloc(sourceSize, sizeof(char));
    pBuffer = (char *)calloc(patternSize, sizeof(char));

	fread( sBuffer, sourceSize, 1, source);
	fread( pBuffer, patternSize, 1, pattern);

	printInfo(sBuffer, pBuffer);

#ifdef CPU
	clock_t begin, end;
	begin = clock();
	boyerMooreInvoke(sBuffer, pBuffer);
	end = clock();
	printf("CPU time: %.6lf s\n", (double)(end - begin)/CLOCKS_PER_SEC);
#endif 

#ifdef GPU
	GPU_boyerMoore(sBuffer, pBuffer);
#endif

	free(sBuffer);
	free(pBuffer);

	return 0;
}









